#include "hip/hip_runtime.h"
//----------------------
// INCLUDES
//----------------------
#include "dct8x8.cuh"
#include "math.h"
#include ""

//----------------------
// IMPLEMENTATION
//----------------------

//*********************************
// Alpha for computing forward 2Dct
//*********************************
__device__ float alpha(int i)
{
    if (i == 0)
        return sqrt(0.125);
    else
        return sqrt(0.25);
}

//***********************************************************
// Compute DCT coef of value (u,v) in matrix
// @param u       : row index
// @param v       : col index
// @param aMatrix : source matrix
// @param offset  : offset of first element 
// @return        : dct coeff for
//***********************************************************
__device__ float computeDCTCoef(int u, int v, float* aMatrix, int offset)
{
    float res = alpha(u)*alpha(v);
    float tmp = 0.0f;

    for (int i = 0; i < ROW_NUMBER; ++i)
    {
        for (int j = 0; j < COL_NUMBER; ++j)
        {
            tmp += cosf(ROW_COEF*u*(2 * i + 1))*cosf(COL_COEF*v*(2 * j + 1))*aMatrix[i * ROW_NUMBER + j];
        }
    }
    return res*tmp;
}

//*************************************************************
// Compute complete DCT of a 8x8 matrix
// See header file for details
//*************************************************************
__global__ void computeDCT2(float* srcMatrixes, float* dstMatrixes, int numberOfMatrixes)
{
    // each thread compute dct for a row
    int threadX = threadIdx.x;

    if (threadX < numberOfMatrixes*ROW_NUMBER)
    {
        int offset = (threadX/ROW_NUMBER)*DCT_MATRIX_SIZE;
        int startIndex = threadX*ROW_NUMBER;
        int u = (startIndex - offset)/ROW_NUMBER;

        for (int v = 0; v < COL_NUMBER; ++v)
        {
            dstMatrixes[startIndex] = computeDCTCoef(u, v, &srcMatrixes[offset], 0);
            ++startIndex;
        }
    }
}

//*******************************************************************
// Wrapper for calling kernel from Cpp source file
// See header file for details
//*******************************************************************
hipError_t wrapperDCT2(float* srcMatrixes, float* dstMatrixes, int numberOfMatrixes)
{
    hipError_t ret = hipSuccess;

    // allocate data on device
    float* devSrcMatrixes;
    float* devDstMatrixes;
    size_t matrixesSize = numberOfMatrixes*DCT_MATRIX_SIZE*sizeof(float);
    ret = hipMalloc(&devSrcMatrixes, matrixesSize);
    if (ret != hipSuccess)
        return ret;

    ret = hipMalloc(&devDstMatrixes, matrixesSize);
    if (ret != hipSuccess)
        return ret;

    // copy source matrixes on device
    ret = hipMemcpy(devSrcMatrixes, srcMatrixes, matrixesSize, hipMemcpyHostToDevice);
    if (ret != hipSuccess)
        return ret;

    // run kernel
    computeDCT2 << <1, DCT_KERNEL_THREADS >> >(devSrcMatrixes, devDstMatrixes, numberOfMatrixes);
    
    // copy destination matrixes back on host
    ret = hipMemcpy(dstMatrixes, devDstMatrixes, matrixesSize, hipMemcpyDeviceToHost);
    if (ret != hipSuccess)
        return ret;

    hipFree(devSrcMatrixes);
    hipFree(devDstMatrixes);
    return ret;
}