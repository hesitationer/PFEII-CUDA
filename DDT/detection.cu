#include "hip/hip_runtime.h"
//---------------------------------------------------
// implementation file of cutImage parallel function
//---------------------------------------------------

//----------------------------
// INCLUDES
//----------------------------
#include "detection.cuh"
#include "kim.cuh"
#include ""
#include "math.h"

//----------------------------
// NAMESPACE
//----------------------------
using namespace utilities;

//----------------------------
// IMPLEMENTATION
//----------------------------

//**********************************************************************
// Parallel version of cutImage. Split given image into image blocks 
// of blockSize^2 this kernel is designed to work with bytes values only
//**********************************************************************
__global__ void cutImageP(unsigned char* imageSrc, ROI imageDims, int blockSize, unsigned char* imageBlocks)
{
    int threadX = threadIdx.x;
    int threadY = threadIdx.y;

    // dimensions of block in image
    int blockWidth = ceilf(imageDims.width / blockSize);
    int blockHeight = ceilf(imageDims.height / blockSize);

    // number of element per image block
    int elemsPerBlocks = blockSize*blockSize;

    // block index
    int blockIndex = threadY + threadX*blockWidth;

    // index on the image the block starts
    int blockStartIndex = threadX*blockWidth*elemsPerBlocks + threadY*blockSize;

    // index(x) on the image block ends
    int blockLimitIndex = blockStartIndex + (blockSize - 1)*imageDims.width;

    // index of the blocks destination buffer
    int imageBlocksIndex;

    // align destination buffer index with index of image block
    imageBlocksIndex = blockIndex*elemsPerBlocks;

    if (threadX < blockHeight && threadY < blockWidth)
    {
        int i, j;

        // fill the image block buffer
        for (i = blockStartIndex; i <= blockLimitIndex; i = i + imageDims.width)
        {
            for (j = i; j < (i + blockSize); ++j)
            {
                imageBlocks[imageBlocksIndex] = imageSrc[j];
                ++imageBlocksIndex;
            }
        }
    }
}

//*******************************************************************
// Wrapper for calling the kernel cutImageP from standard source file
//*******************************************************************
hipError_t cutImagePWrapper(unsigned char* image, const ROI& imageDims, int blockSize, unsigned char* imageBlocks)
{
    hipError_t ret = hipSuccess;

    size_t imageSize = (size_t)(imageDims.width*imageDims.height);

    uchar* devImageData;
    uchar* devImageBlocks;

    ret = hipMalloc(&devImageData, imageSize);

    if (ret != hipSuccess)
    {
        printf("Error while allocating devImageData on device\n");
        return ret;
    }

    ret = hipMalloc(&devImageBlocks, imageSize);

    if (ret != hipSuccess)
    {
        printf("Error while allocating devImageBlocks on device\n");
        return ret;
    }

    // copy data on device
    ret = hipMemcpy(devImageData, image, imageSize, hipMemcpyHostToDevice);

    if (ret != hipSuccess)
    {
        printf("Error while copying devImageData on device\n");
        return ret;
    }

    // run kernel
    dim3 threadsDim(CUT_IMG_KERNEL_THREADS, CUT_IMG_KERNEL_THREADS, 1);
    cutImageP << <1, threadsDim >> >(devImageData, imageDims, blockSize, devImageBlocks);

    // copy data back on host
    ret = hipMemcpy(imageBlocks, devImageBlocks, imageSize, hipMemcpyDeviceToHost);

    if (ret != hipSuccess)
    {
        printf("Error while copying devImageBlocks on host\n");
        return ret;
    }
    return ret;
}


//**********************************************************************
// Parallel version of a part of processing of OnlineDetection algorithm
// See header file for details
//**********************************************************************
__global__ void markImageDefectsP(float alpha, float* srcDistMatrix, size_t distMatrixSize, ROI distMatrixDims, int blockSize, unsigned char* imageBlocks, unsigned char* markedImage, ROI imageDims)
{
    // thread index, used to index the block which will be processed
    int threadX = threadIdx.x;
    int threadY = threadIdx.y;

    // image indexes x-y
    int imageX = threadX*blockSize;
    int imageY = threadY*blockSize;

    // image index one dimension
    int imageIndex = imageX*imageDims.width + imageY;

    // distance matrix index and size, given by how many blocks
    int distIndex = threadX*distMatrixDims.width + threadY;

    // number of elements per blocks
    int elemPerBlocks = blockSize*blockSize;

    // block index
    int blockIndex = distIndex*elemPerBlocks;

    // image index of the last line of a block
    int imageIndexTo = imageIndex + (blockSize - 1)*imageDims.width;

    if (threadX < distMatrixDims.height && threadY < distMatrixDims.width)
    {
        float distance = srcDistMatrix[distIndex];

        // block is considered defectious, should copy it on 
        // destination iimage
        if (distance > alpha)
        {
            int i, j;

            for (i = imageIndex; i <= imageIndexTo; i = i + imageDims.width)
            {
                for (j = i; j < i + blockSize; ++j)
                {
                    unsigned char pixelValue = imageBlocks[blockIndex];
                    markedImage[j] = pixelValue;
                    ++blockIndex;
                }
            }
        }
    }
}

//***************************************************************************
// Wrapper for calling the kernel markImageDefectsP from standard source file
//***************************************************************************
hipError_t markImageDefectsPWrapper(float alpha, float* srcDistMatrix, int blockSize, ROI imgBlockDims, unsigned char* imageBlocks, unsigned char* markedImage, ROI imageDims)
{
    hipError_t ret;

    // data allocation on device
    float *devDistMatrix;
    size_t distMatrixSize = (size_t)(imgBlockDims.width*imgBlockDims.height*sizeof(float));

    unsigned char *devImageBlocks, *devMarkedImage;
    size_t imageSize = (size_t)(imageDims.width*imageDims.height);

    // distance matrix
    ret = hipMalloc(&devDistMatrix, distMatrixSize);
    if (ret != hipSuccess)
    {
        return ret;
    }

    // image blocks
    ret = hipMalloc(&devImageBlocks, imageSize);
    if (ret != hipSuccess)
    {
        return ret;
    }

    // marked image buffer
    ret = hipMalloc(&devMarkedImage, imageSize);
    if (ret != hipSuccess)
    {
        return ret;
    }

    // data copy on device
    // distance matrix
    ret = hipMemcpy(devDistMatrix, srcDistMatrix, distMatrixSize, hipMemcpyHostToDevice);
    if (ret != hipSuccess)
    {
        return ret;
    }

    // image blocks
    ret = hipMemcpy(devImageBlocks, imageBlocks, imageSize, hipMemcpyHostToDevice);
    if (ret != hipSuccess)
    {
        return ret;
    }

    // matrix size in dimension, was in byte for malloc
    distMatrixSize /= sizeof(float);
    dim3 threadsDim(MARK_IMG_KERNEL_THREADS, MARK_IMG_KERNEL_THREADS, 1);
    markImageDefectsP << <1, threadsDim >> >(alpha, devDistMatrix, distMatrixSize, imgBlockDims, blockSize, devImageBlocks, devMarkedImage, imageDims);

    // data copy back to host
    ret = hipMemcpy(markedImage, devMarkedImage, imageSize, hipMemcpyDeviceToHost);

    hipFree(devDistMatrix);
    hipFree(devImageBlocks);
    hipFree(devMarkedImage);
    return ret;
}

__global__ void computeEuclidianDistanceP(float* refSign, float* signs, int numberOfsigns, float* distances, float* distanceMean)
{
    int signIndex = threadIdx.x;
    float distance = 0.0f;

    if (signIndex < numberOfsigns)
    {
        for (int i = 0; i < KIM_SIGN_SIZE; i++)
        {
            distance += fabsf(refSign[i] - signs[signIndex*KIM_SIGN_SIZE + i]);
        }
        distances[signIndex] = distance;
        *distanceMean += distance;
    }
}

//***********************************************************************************
// Wrapper for calling the kernel computeEuclidianDistanceP from standard source file
//***********************************************************************************
hipError_t computeEuclidianDistancePWrapper(float* refSign, float* signs, int numberOfsigns, float* distances, float* distanceMean)
{
    hipError_t ret = hipSuccess;
    // allocate data on device
    float* devRefSign;
    float* devSigns;
    float* devDistances;
    float* devDistanceMean;

    size_t refSignSize = KIM_SIGN_SIZE*sizeof(float);
    size_t signsSize = KIM_SIGN_SIZE*numberOfsigns*sizeof(float);
    size_t distancesSize = numberOfsigns*sizeof(float);

    ret = hipMalloc(&devRefSign, refSignSize);
    if (ret != hipSuccess)
        return ret;

    ret = hipMalloc(&devSigns, signsSize);
    if (ret != hipSuccess)
        return ret;

    ret = hipMalloc(&devDistances, distancesSize);
    if (ret != hipSuccess)
        return ret;

    ret = hipMalloc(&devDistanceMean, sizeof(float));
    if (ret != hipSuccess)
        return ret;

    // copy refSignature and Signatures to device
    ret = hipMemcpy(devRefSign, refSign, refSignSize, hipMemcpyHostToDevice);
    if (ret != hipSuccess)
        return ret;

    ret = hipMemcpy(devSigns, signs, signsSize, hipMemcpyHostToDevice);
    if (ret != hipSuccess)
        return ret;

    // run kernel
    computeEuclidianDistanceP << <1, CALC_DISTANCE_KERNEL_THREADS >> >(devRefSign, devSigns, numberOfsigns, devDistances, devDistanceMean);

    // copy data back on host
    ret = hipMemcpy(distances, devDistances, distancesSize, hipMemcpyDeviceToHost);
    if (ret != hipSuccess)
        return ret;

    ret = hipMemcpy(distanceMean, devDistanceMean, sizeof(float), hipMemcpyDeviceToHost);
    if (ret != hipSuccess)
        return ret;

    // deallocate data on device
    hipFree(devDistances);
    hipFree(devRefSign);
    hipFree(devSigns);
    hipFree(devDistanceMean);
    return ret;
}