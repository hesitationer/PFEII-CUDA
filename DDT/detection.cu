#include "hip/hip_runtime.h"
//---------------------------------------------------
// implementation file of cutImage parallel function
//---------------------------------------------------

//----------------------------
// INCLUDES
//----------------------------
#include "detection.cuh"
#include "kim.cuh"
#include "dct8x8.cuh"
#include ""
#include "math.h"

//----------------------------
// NAMESPACE
//----------------------------
using namespace utilities;
using namespace std;

//----------------------------
// IMPLEMENTATION
//----------------------------

//**********************************************************************
// Parallel version of cutImage. Split given image into image blocks 
// of blockSize^2 this kernel is designed to work with bytes values only
//**********************************************************************
__global__ void cutImageP(unsigned char* imageSrc, ROI imageDims, int blockSize, unsigned char* imageBlocks)
{
    int threadX = threadIdx.x;
    int threadY = threadIdx.y;

    // dimensions of block in image
    int blockWidth = ceilf(imageDims.width / blockSize);
    int blockHeight = ceilf(imageDims.height / blockSize);

    // number of element per image block
    int elemsPerBlocks = blockSize*blockSize;

    // block index
    int blockIndex = threadY + threadX*blockWidth;

    // index on the image the block starts
    int blockStartIndex = threadX*blockWidth*elemsPerBlocks + threadY*blockSize;

    // index(x) on the image block ends
    int blockLimitIndex = blockStartIndex + (blockSize - 1)*imageDims.width;

    // index of the blocks destination buffer
    int imageBlocksIndex;

    // align destination buffer index with index of image block
    imageBlocksIndex = blockIndex*elemsPerBlocks;

    if (threadX < blockHeight && threadY < blockWidth)
    {
        int i, j;

        // fill the image block buffer
        for (i = blockStartIndex; i <= blockLimitIndex; i = i + imageDims.width)
        {
            for (j = i; j < (i + blockSize); ++j)
            {
                imageBlocks[imageBlocksIndex] = imageSrc[j];
                ++imageBlocksIndex;
            }
        }
    }
}

//*******************************************************************
// Wrapper for calling the kernel cutImageP from standard source file
//*******************************************************************
hipError_t cutImagePWrapper(unsigned char* image, const ROI& imageDims, int blockSize, unsigned char* imageBlocks)
{
    hipError_t ret = hipSuccess;

    size_t imageSize = (size_t)(imageDims.width*imageDims.height);

    uchar* devImageData;
    uchar* devImageBlocks;

    ret = hipMalloc(&devImageData, imageSize);

    if (ret != hipSuccess)
    {
        printf("Error while allocating devImageData on device\n");
        return ret;
    }

    ret = hipMalloc(&devImageBlocks, imageSize);

    if (ret != hipSuccess)
    {
        printf("Error while allocating devImageBlocks on device\n");
        return ret;
    }

    // copy data on device
    ret = hipMemcpy(devImageData, image, imageSize, hipMemcpyHostToDevice);

    if (ret != hipSuccess)
    {
        printf("Error while copying devImageData on device\n");
        return ret;
    }

    // run kernel
    dim3 threadsDim(CUT_IMG_KERNEL_THREADS, CUT_IMG_KERNEL_THREADS, 1);
    cutImageP << <1, threadsDim >> >(devImageData, imageDims, blockSize, devImageBlocks);

    // copy data back on host
    ret = hipMemcpy(imageBlocks, devImageBlocks, imageSize, hipMemcpyDeviceToHost);

    if (ret != hipSuccess)
    {
        printf("Error while copying devImageBlocks on host\n");
        return ret;
    }
    return ret;
}


//**********************************************************************
// Parallel version of a part of processing of OnlineDetection algorithm
// See header file for details
//**********************************************************************
__global__ void markImageDefectsP(float alpha, float* srcDistMatrix, size_t distMatrixSize, ROI distMatrixDims, int blockSize, unsigned char* imageBlocks, unsigned char* markedImage, ROI imageDims)
{
    // thread index, used to index the block which will be processed
    int threadX = threadIdx.x;
    int threadY = threadIdx.y;

    // image indexes x-y
    int imageX = threadX*blockSize;
    int imageY = threadY*blockSize;

    // image index one dimension
    int imageIndex = imageX*imageDims.width + imageY;

    // distance matrix index and size, given by how many blocks
    int distIndex = threadX*distMatrixDims.width + threadY;

    // number of elements per blocks
    int elemPerBlocks = blockSize*blockSize;

    // block index
    int blockIndex = distIndex*elemPerBlocks;

    // image index of the last line of a block
    int imageIndexTo = imageIndex + (blockSize - 1)*imageDims.width;

    if (threadX < distMatrixDims.height && threadY < distMatrixDims.width)
    {
        float distance = srcDistMatrix[distIndex];

        // block is considered defectious, should copy it on 
        // destination iimage
        if (distance > alpha)
        {
            int i, j;

            for (i = imageIndex; i <= imageIndexTo; i = i + imageDims.width)
            {
                for (j = i; j < i + blockSize; ++j)
                {
                    unsigned char pixelValue = imageBlocks[blockIndex];
                    markedImage[j] = pixelValue;
                    ++blockIndex;
                }
            }
        }
    }
}

//***************************************************************************
// Wrapper for calling the kernel markImageDefectsP from standard source file
//***************************************************************************
hipError_t markImageDefectsPWrapper(float alpha, float* srcDistMatrix, int blockSize, ROI imgBlockDims, unsigned char* imageBlocks, unsigned char* markedImage, ROI imageDims)
{
    hipError_t ret;

    // data allocation on device
    float *devDistMatrix;
    size_t distMatrixSize = (size_t)(imgBlockDims.width*imgBlockDims.height*sizeof(float));

    unsigned char* devImageBlocks;
    unsigned char* devMarkedImage;
    size_t imageSize = (size_t)(imageDims.width*imageDims.height);

    // distance matrix
    ret = hipMalloc(&devDistMatrix, distMatrixSize);
    if (ret != hipSuccess)
    {
        return ret;
    }

    // image blocks
    ret = hipMalloc(&devImageBlocks, imageSize);
    if (ret != hipSuccess)
    {
        return ret;
    }

    // marked image buffer
    ret = hipMalloc(&devMarkedImage, imageSize);
    if (ret != hipSuccess)
    {
        return ret;
    }

    // data copy on device
    // distance matrix
    ret = hipMemcpy(devDistMatrix, srcDistMatrix, distMatrixSize, hipMemcpyHostToDevice);
    if (ret != hipSuccess)
    {
        return ret;
    }

    // image blocks
    ret = hipMemcpy(devImageBlocks, imageBlocks, imageSize, hipMemcpyHostToDevice);
    if (ret != hipSuccess)
    {
        return ret;
    }

    // matrix size in dimension, was in byte for malloc
    distMatrixSize /= sizeof(float);
    dim3 threadsDim(MARK_IMG_KERNEL_THREADS, MARK_IMG_KERNEL_THREADS, 1);
    markImageDefectsP << <1, threadsDim >> >(alpha, devDistMatrix, distMatrixSize, imgBlockDims, blockSize, devImageBlocks, devMarkedImage, imageDims);

    // data copy back to host
    ret = hipMemcpy(markedImage, devMarkedImage, imageSize, hipMemcpyDeviceToHost);

    hipFree(devDistMatrix);
    hipFree(devImageBlocks);
    hipFree(devMarkedImage);
    return ret;
}

__global__ void computeEuclidianDistanceP(float* refSign, float* signs, int numberOfsigns, float* distances, float* distanceMean)
{
    int signIndex = threadIdx.x;
    float distance = 0.0f;

    if (signIndex < numberOfsigns)
    {
        for (int i = 0; i < KIM_SIGN_SIZE; i++)
        {
            distance += fabsf(refSign[i] - signs[signIndex*KIM_SIGN_SIZE + i]);
        }
        distances[signIndex] = distance;
        *distanceMean += distance;
    }
}

//***********************************************************************************
// Wrapper for calling the kernel computeEuclidianDistanceP from standard source file
//***********************************************************************************
hipError_t computeEuclidianDistancePWrapper(float* refSign, float* signs, int numberOfsigns, float* distances, float* distanceMean)
{
    hipError_t ret = hipSuccess;
    // allocate data on device
    float* devRefSign;
    float* devSigns;
    float* devDistances;
    float* devDistanceMean;

    size_t refSignSize = KIM_SIGN_SIZE*sizeof(float);
    size_t signsSize = KIM_SIGN_SIZE*numberOfsigns*sizeof(float);
    size_t distancesSize = numberOfsigns*sizeof(float);

    ret = hipMalloc(&devRefSign, refSignSize);
    if (ret != hipSuccess)
        return ret;

    ret = hipMalloc(&devSigns, signsSize);
    if (ret != hipSuccess)
        return ret;

    ret = hipMalloc(&devDistances, distancesSize);
    if (ret != hipSuccess)
        return ret;

    ret = hipMalloc(&devDistanceMean, sizeof(float));
    if (ret != hipSuccess)
        return ret;

    // copy refSignature and Signatures to device
    ret = hipMemcpy(devRefSign, refSign, refSignSize, hipMemcpyHostToDevice);
    if (ret != hipSuccess)
        return ret;

    ret = hipMemcpy(devSigns, signs, signsSize, hipMemcpyHostToDevice);
    if (ret != hipSuccess)
        return ret;

    // run kernel
    computeEuclidianDistanceP << <1, CALC_DISTANCE_KERNEL_THREADS >> >(devRefSign, devSigns, numberOfsigns, devDistances, devDistanceMean);

    // copy data back on host
    ret = hipMemcpy(distances, devDistances, distancesSize, hipMemcpyDeviceToHost);
    if (ret != hipSuccess)
        return ret;

    ret = hipMemcpy(distanceMean, devDistanceMean, sizeof(float), hipMemcpyDeviceToHost);
    if (ret != hipSuccess)
        return ret;

    // deallocate data on device
    hipFree(devDistances);
    hipFree(devRefSign);
    hipFree(devSigns);
    hipFree(devDistanceMean);
    return ret;
}

hipError_t detectionWrapperP(unsigned char* image, ROI imageDims, int blockSize, ROI imageBlocksDims, const vector<float>& refSign, float eta, unsigned char* markedImage)
{
    hipError_t ret = hipSuccess;
    int imageSize = imageDims.width*imageDims.height;
    int imageblocksCount = imageBlocksDims.width*imageBlocksDims.height;
    
    // 1. call cut image kernel
    // image alloc
    unsigned char* devImage;
    size_t imageSizeMem = (size_t)imageSize;
    ret = hipMalloc(&devImage, imageSizeMem);
    ret = hipMemcpy(devImage, image, imageSizeMem, hipMemcpyHostToDevice);

    // image blocks alloc
    unsigned char* devImageBlocks;
    ret = hipMalloc(&devImageBlocks, imageSizeMem);

    // run kernel
    dim3 threadsDim(CUT_IMG_KERNEL_THREADS, CUT_IMG_KERNEL_THREADS, 1);
    cutImageP << <1, threadsDim >> >(devImage, imageDims, blockSize, devImageBlocks);

    // 2. kim signature phase 1
    // mean matrixes
    float* devMeanMatrixes;
    size_t meanMatrixesSizeMem = imageblocksCount*MEAN_MATRIX_SIZE*sizeof(float);
    ret = hipMalloc(&devMeanMatrixes, meanMatrixesSizeMem);

    // run kernel
    threadsDim.x = PH1_KERNEL_THREADS;
    threadsDim.y = PH1_KERNEL_THREADS;
    extractKimSignaturePhase1 << <1, threadsDim >> >(devImageBlocks, imageBlocksDims, blockSize, devMeanMatrixes);

    // 3. kim signature phase 2 (DCT on mean matrix)
    // dct matrixes alloc
    float *devDctMeanMatrixes;
    ret = hipMalloc(&devDctMeanMatrixes, meanMatrixesSizeMem);

    // run kernel
    computeDCT2 << <1, DCT_KERNEL_THREADS >> >(devMeanMatrixes, devDctMeanMatrixes, imageblocksCount);

    // 4. kim signature phase 3
    // kim signatures
    float* devKimSignatures;
    size_t kimSignatureSizeMem = imageblocksCount*KIM_SIGN_SIZE*sizeof(float);
    ret = hipMalloc(&devKimSignatures, kimSignatureSizeMem);

    extractKimSignaturePhase3<< <1, PH3_KERNEL_THREADS >> >(devDctMeanMatrixes, imageblocksCount, devKimSignatures);

    // 5. distance euclidian distance compute
    // ref signature change from vector -> float*
    float pRefSign[KIM_SIGN_SIZE];
    float distanceMean = 0.0;
    std::copy(refSign.begin(), refSign.end(), pRefSign);

    // allocate on device
    float* devRefSign;
    size_t refSignSizeMem = KIM_SIGN_SIZE*sizeof(float);
    ret = hipMalloc(&devRefSign, refSignSizeMem);
    ret = hipMemcpy(devRefSign, pRefSign, refSignSizeMem, hipMemcpyHostToDevice);

    // distances matrixe
    float* hostDistances = new float[imageblocksCount];
    float* devDistanceMatrixes;
    size_t distancesSizeMem = imageblocksCount*sizeof(float);
    ret = hipMalloc(&devDistanceMatrixes, distancesSizeMem);
    
    // mean distance holder
    float* devDistanceMean;
    ret = hipMalloc(&devDistanceMean, sizeof(float));

    computeEuclidianDistanceP << <1, CALC_DISTANCE_KERNEL_THREADS >> >(devRefSign, devKimSignatures, imageblocksCount, devDistanceMatrixes, devDistanceMean);

    // copy back some needed data on host
    ret = hipMemcpy(hostDistances, devDistanceMatrixes, distancesSizeMem, hipMemcpyDeviceToHost);
    if (ret != hipSuccess) return ret;

    ret = hipMemcpy(&distanceMean, devDistanceMean, sizeof(float), hipMemcpyDeviceToHost);
    if (ret != hipSuccess) return ret;

    // 6. compute alpha treshold
    distanceMean = distanceMean / (float)imageblocksCount;
    vector<float> distanceVec(imageblocksCount);
    std::copy(hostDistances, hostDistances + imageblocksCount, distanceVec.begin());

    float alpha = distanceMean + (eta*iqr<float>(distanceVec));

    // 7. mark defects on image
    unsigned char* devMarkedImage;
    size_t markedImageSizeMem = (size_t)imageSize;
    ret = hipMalloc(&devMarkedImage, markedImageSizeMem);

    int distMatrixSize = distancesSizeMem /sizeof(float);
    threadsDim.x = MARK_IMG_KERNEL_THREADS;
    threadsDim.y = MARK_IMG_KERNEL_THREADS;
    markImageDefectsP << < 1, threadsDim >> >(alpha, devDistanceMatrixes, distMatrixSize, imageBlocksDims, blockSize, devImageBlocks, devMarkedImage, imageDims);

    // copy final data back to host
    ret = hipMemcpy(markedImage, devMarkedImage, imageSize, hipMemcpyDeviceToHost);

    // and clean all device memory allocated
    hipFree(devImage);
    hipFree(devImageBlocks);
    hipFree(devMeanMatrixes);
    hipFree(devDctMeanMatrixes);
    hipFree(devKimSignatures);
    hipFree(devRefSign);
    hipFree(devDistanceMatrixes);
    hipFree(devDistanceMean);
    hipFree(devMarkedImage);

    //delete[] pRefSign;
    delete[] hostDistances;
    return ret;
}